
#include <hip/hip_runtime.h>
#define TILE_SIZE 26

__global__ void MatMulKernel(float* A, float* B, float* C, int N){
	
	int iudir = 0, iuid = 0, iu =0;
	asm("//lol");
	int i = 0,j = 0;
	int iters = N/TILE_SIZE;
	int num_timeSteps = gridDim.x+gridDim.y+iters-1;
	int start_timeStep = blockIdx.x+blockIdx.y+1;
	int A_start_row, A_start_col, B_start_row, B_start_col;


	 __shared__ float cb_A[TILE_SIZE][TILE_SIZE];
	 __shared__ float cb_B[TILE_SIZE][TILE_SIZE];


	//Initialize output array in shared mem

	float c = 0;

	asm("// iu");
	if(iu){
		asm("// iudir");
	  if(iudir==2){
	  	asm("// iuid");
	  	A_start_row = iuid * TILE_SIZE;
	  }  
	  asm("// iudir");
	  if(iudir==0){
	  	asm("// iuid");
	  	B_start_col = iuid * TILE_SIZE;
	  } 
	}
	else{
	  A_start_row = blockIdx.y * TILE_SIZE;
	  B_start_col = blockIdx.x * TILE_SIZE;
	}
	i = 0;

	for(int time_step=0; time_step<num_timeSteps; time_step++){

	  A_start_col = i*TILE_SIZE;
	  B_start_row = A_start_col;
	  asm("// iu");
	  if(iu){
	  	asm("// iudir");
		if(iudir==2){
			asm("// iuid");
			if((time_step>=iuid)&&(i<iters)){
				cb_A[threadIdx.y][threadIdx.x] = A[(A_start_row + threadIdx.y)*N+(A_start_col + threadIdx.x)];
				i++;
	  		}

		}

	    asm("// iudir");
	    if(iudir==0){
	    	asm("// iuid");
			if((time_step>=iuid)&&(i<iters)){
				cb_B[threadIdx.y][threadIdx.x] = B[(B_start_row + threadIdx.y)*N+(B_start_col + threadIdx.x)];
			   	i++;
			}
		}

	  }

	  else{
		if((start_timeStep <= time_step) && (i<iters)){
			

		    //Accumulate matrix product for the current input tile in shared mem

		    for(j=0; j<TILE_SIZE; j++)
		    {
		        c += cb_A[threadIdx.y][j] * cb_B[j][threadIdx.x];

		    }

		    __syncthreads();

		    //Load the tile of A in shared mem to the south CB
		    cb_A[threadIdx.y][threadIdx.x] = cb_A[threadIdx.y][threadIdx.x];
		    cb_B[threadIdx.y][threadIdx.x] = cb_B[threadIdx.y][threadIdx.x];
			
		    i++;
		}

	  }

	  asm("bar.blocksync	0;");

	 
	}

	asm("// iu");
	if(!iu){
	  //Copy computed product tile to global mem
	  C[(A_start_row+threadIdx.y)*N + (B_start_col+threadIdx.x)] = c;
	}
}