#include "hip/hip_runtime.h"
/***********************************************
	Matrix Multiplication ver 1.0

Host code for floating point matrix mult
with array for storing timestep of each block's
starting iteration.

Author: Revathy Rajasree
************************************************/


//Headers

#include <stdio.h>
#include "matmult.h"

//Macro for CUDA error handling
#define CUDA_CHECK_RETURN(value) {   \
    hipError_t _m_cudaStat = value;  \
    if (_m_cudaStat != hipSuccess) {  \
      fprintf(stderr, "Error: %s at line %d in file %s\n",hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
      exit(1);     \
    } \
}


//Host arrays
float *h_A, *h_B, *h_C;

int *h_blkStart;

//Device arrays
float *d_A, *d_B, *d_C, *g_CB;

int *d_blkStart;

//function declarations
void fillMatrix(float*, int, int);
void fillIdentity(float*, int);
void printMatrix(float*, int, int);
void matMult(float*, float*, float*, int, int, int);
void matMultVerify(float*, float*, float*, int, int, int);
void Cleanup();

//Host code
int main(int argc, char** argv)
{

    int N;

    //Read the arguments
    if(argc <= 1){
	printf("Error: Expected matrix size input.\n");
	exit(0);
    }
    else{
	sscanf(argv[1], "%d", &N);
    }

    N = INPUT_SIZE;

    printf("MATRIX SIZE = %d\n", N);
    printf("TILE SIZE = %d \n", TILE_SIZE);
    printf("\n");

    //Allocate host memory

    h_A = (float *) malloc(N*N*sizeof(float));

    h_B = (float *) malloc(N*N*sizeof(float));

    h_C = (float *) malloc(N*N*sizeof(float));

    //Initialize the input matrices
    fillMatrix(h_A, N, N);
    fillMatrix(h_B, N, N);
    //fillIdentity(h_B, N);
    
    //Allocate memory on device
    //hipError_t error;
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_A, N*N*sizeof(float)));
    
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_B, N*N*sizeof(float)));

    CUDA_CHECK_RETURN(hipMalloc((void**)&d_C, N*N*sizeof(float)));

    CUDA_CHECK_RETURN(hipMalloc((void**)&g_CB, 4096*8*NUMBER_OF_SPILLS));



    //Copy inputs to device
    CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, N*N*sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_B, h_B, N*N*sizeof(float), hipMemcpyHostToDevice));

    //Compute kernel launch parameters
    int numBlocks, numThreads;

    numThreads = TILE_SIZE;
    numBlocks  = N/TILE_SIZE;

    //Allocate array on device array to record block's starting timestep
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_blkStart, numBlocks*numBlocks*sizeof(int)));

/*
    if(N % TILE_SIZE == 0) numBlocks = N/TILE_SIZE;
    else numBlocks = (N+TILE_SIZE)/TILE_SIZE;
*/

    printf("Number of blocks  = %d x %d \n", numBlocks, numBlocks);
    printf("Threads per block = %d x %d \n", numThreads, numThreads);


    dim3 dimGrid(numBlocks, numBlocks);
    dim3 dimBlock(numThreads, numThreads); 

    //Launch matrix multiplication kernel on GPU
    matMultiply <<<dimGrid, dimBlock>>> (d_A, d_B, d_C, N, TILE_SIZE, d_blkStart, g_CB);
 
    CUDA_CHECK_RETURN(hipGetLastError()); 

    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    //copy result from device
    CUDA_CHECK_RETURN(hipMemcpy(h_C, d_C, N*N*sizeof(float), hipMemcpyDeviceToHost));

    //verify result with CPU 
    matMultVerify(h_A, h_B, h_C, N, N, N);

    // float *C;
    // C = (float *) malloc(N*N*sizeof(float));

    // matMult(h_A, h_B, C, N, N, N);

    /*
    //Print the matrix
    printf("Input matrix A: \n");
    printMatrix(h_A, N, N);

    printf("\nInput matrix B: \n");
    printMatrix(h_B, N, N);


    //Matrix multiplication on CPU
    matMult(h_A, h_B, h_C, N, N, N);

    printf("\nProduct matrix CPU: \n");
    printMatrix(C, N, N);

    */
   //  printf("\nProduct matrix GPU:\n");
   //  printMatrix(h_C, N, N);


   // int i,j;

 //    fprintf(stderr, "\nProduct CPU :: GPU  \n");
 //    for(i=0; i<N; i++){
	// for(j=0; j<N; j++){
	    
	//    fprintf(stderr, "%7.3f %7.3f \n",C[i*N + j], h_C[i*N + j]);

	// }
	// fprintf(stderr,"\n");

 //    }



//    free(C);


    //allocate array on host to hold threadblocks' starting timestep
    // h_blkStart = (int *) malloc(numBlocks*numBlocks*sizeof(int));

    // //copy the recorded timesteps to host
    // CUDA_CHECK_RETURN(hipMemcpy(h_blkStart, d_blkStart, numBlocks*numBlocks*sizeof(int), hipMemcpyDeviceToHost));

 //    //Display each blocks starting timestep
 //    printf("\n Blocks' start schedule:  \n");

 //    for(i=0; i<numBlocks; i++)
 //    {
	// for(j=0; j<numBlocks; j++)
	// {
	//     printf("%4d  ", h_blkStart[i*numBlocks + j]);

	// }
	// printf("\n");
 //    }


    Cleanup();
    return 0;
}

void Cleanup()
{
    //Free host memory
    if(h_A) free(h_A);
    if(h_B) free(h_B);
    if(h_C) free(h_C);
    if(h_blkStart) free(h_blkStart);

    //free device memory
    if(d_A) hipFree(d_A);
    if(d_B) hipFree(d_B);
    if(d_C) hipFree(d_C);
    if(d_blkStart) hipFree(d_blkStart);

    return;

}

void matMultVerify(float* A, float* B, float* C, int rA, int n, int cB)
{
    int i,j,k;
    float temp;

    for(i=0; i<rA; i++){
        for(j=0; j<cB; j++){

            temp = 0.0;
            for(k=0; k<n; k++){

                temp += A[i*n+k] * B[k*cB+j];
            }
	
	    if (fabs(temp - C[i*cB+j]) > 1e-3){
		printf("Result error: At (%d, %d), CPU val = %f; GPU val = %f \n", i,j,temp,C[i*cB+j] );
		printf("TEST FAILED. \n");
		return;
	    }
        }

    }

    printf("TEST PASSED. \n");
    return;
}


//Matrix multiplication on CPU

void matMult(float* A, float* B, float* C, int rA, int n, int cB)
{
    int i,j,k;

    for(i=0; i<rA; i++){
	for(j=0; j<cB; j++){

	    C[i*cB+j]=0;
	    for(k=0; k<n; k++){
		
		C[i*cB+j] += A[i*n+k] * B[k*cB+j];
	    }
	}

    }

    return;

}

//Function that fills a matrix with real values

void fillMatrix(float* mat, int row_size, int col_size)
{
    int i,j;

    for(i=0; i<row_size; i++)
    {
	for(j=0; j<col_size; j++)
	{
		//mat[i*col_size + j] = ((float(i+1)) *(float)(j+1))/init ;
		mat[i*col_size + j] = ((float)rand())/RAND_MAX ; 
	}

    }
    return;
}

//Function that fills a square matrix of size NxN with identity mtrix

void fillIdentity(float* mat, int N)
{
    int i,j;

    for(i=0; i<N; i++){
	for(j=0; j<N; j++){
		
		if(i==j) mat[i*N + j] = 1.0;
		else     mat[i*N + j] = 0.0;
    	}

    }
    return;
}

//Function to print a matrix

void printMatrix(float* mat, int row_size, int col_size)
{
    int i, j;

    for(i=0; i<row_size; i++){
    	for(j=0; j<col_size; j++){
            // if(mat[i*col_size + j] != 0){
            //     fprintf(stderr, "1");
            // }
            // else{
            //     fprintf(stderr, "0");
            // }
    		fprintf(stderr,"%1.0f", mat[i*col_size + j]);
    	}
        fprintf(stderr, "\n");
    }
}

