#include "hip/hip_runtime.h"

#include "matmult.h"

//Device Kernel to compute matrix product
//with pipelined block execution


__global__ void matMultiply (float* A, float* B, float* C, int N, int b, int* blk_start, float* g_CB)
{

    __shared__ float s_A[TILE_SIZE][TILE_SIZE];
    __shared__ float s_B[TILE_SIZE][TILE_SIZE];
    __shared__ float s_C[TILE_SIZE][TILE_SIZE];


    int i,j,t;
    int iters = N/TILE_SIZE;
    int num_timeSteps = gridDim.x+gridDim.y+iters-2;
    int start_timeStep = blockIdx.x+blockIdx.y;
    int blk_start_idx = blockIdx.y * gridDim.x + blockIdx.x;

    int A_start_row, A_start_col, B_start_row, B_start_col;

    //Initialize output array in shared mem

    s_C[threadIdx.y][threadIdx.x] = 0;

    A_start_row = blockIdx.y * TILE_SIZE;
    B_start_col = blockIdx.x * TILE_SIZE;

    i = 0;

    for(t=0; t<num_timeSteps; t++)
    {
	if((start_timeStep <=t) && (i<iters))
	{

	    //Record 't' of first iteration
	    if((threadIdx.y*blockDim.x+threadIdx.x ==0) && (i==0))
		blk_start[blk_start_idx] = t;

	    A_start_col = i*TILE_SIZE;
	    B_start_row = A_start_col;

	    //Copy a tile of A to shared mem
	    s_A[threadIdx.y][threadIdx.x] = A[(A_start_row + threadIdx.y)*N+(A_start_col + threadIdx.x)];


	    //Copy corresponding tile of B to shared mem
	    s_B[threadIdx.y][threadIdx.x] = B[(B_start_row + threadIdx.y)*N+(B_start_col + threadIdx.x)];


	    __syncthreads();

	    //Accumulate matrix product for the current input tile in shared mem

	    for(j=0; j<TILE_SIZE; j++)
	    {
	        s_C[threadIdx.y][threadIdx.x] += s_A[threadIdx.y][j] * s_B[j][threadIdx.x];

	    }

	    __syncthreads();

	    i++;
	}
    }


    //Copy computed product tile to global mem
    C[(A_start_row+threadIdx.y)*N + (B_start_col+threadIdx.x)] = s_C[threadIdx.y][threadIdx.x];
    
}
